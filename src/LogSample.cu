#include "hip/hip_runtime.h"
#include "THC.h"
#include "utils.h"
#include "luaT.h"

#include <lua.h>
#include <lauxlib.h>
#include <lualib.h>

#include "THCTensorRandom.h"
#include "THCDeviceUtils.cuh"
#include "THCReduceApplyUtils.cuh"
#include "THCApply.cuh"
#include "THCGeneral.h"
#include "THCTensorCopy.h"
#include "THCTensorMath.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_mtgp32_host.h>
#include <rocrand/rocrand_mtgp32_11213.h>
#include <hip/hip_math_constants.h>

#define MAX_NUM_BLOCKS 64
#define BLOCK_SIZE 256

__global__ void
sampleLogMultinomialOnce(float* dest,
                         long distributions,
                         int categories,
                         float* dist) {
  extern __shared__ float smem[];
  
  for (long curDist = blockIdx.x; curDist < distributions; curDist += gridDim.x) {
    
    // Each block handles one distribution
    // First pass, find the total sum of the distribution
    float sum = -HIP_INF_F;
    for (int cat = threadIdx.x; cat < categories; cat += blockDim.x) {
      sum = device_log_add(sum, dist[curDist * categories + cat]); // 1d indexing into 2d array
    }
    
    // threadIdx.x == 0 has the sum value from this
    sum = reduceBlock(smem, blockDim.x, sum, device_log_add_functor(), -HIP_INF_F);
    
    // Broadcast sum and sample value
    if (threadIdx.x == 0) {
      smem[0] = sum;
      smem[1] = log(dest[curDist]) + sum;
    }
    __syncthreads();
    
    sum = smem[0];
    float sample = smem[1];
    __syncthreads();
    
    if (sum == -HIP_INF_F || sample == -HIP_INF_F) {
      // Choose the first element
      if (threadIdx.x == 0) {
        dest[curDist] = 1;
      }
      
      continue;
    }
    
    int chunks = THCCeilDiv(categories, (int) blockDim.x);
    float prevHighProb = -HIP_INF_F;
    
    for (int chunk = 0; chunk < chunks; ++chunk) {
      // All threads in bounds load a value
      int cat = chunk * blockDim.x + threadIdx.x;
      
      float val = cat < categories ? dist[curDist * categories + cat] : -HIP_INF_F;
      smem[threadIdx.x] = val;
      __syncthreads();
      
      // Perform an inclusive prefix sum of the shared memory contents
      for (int offset = 1; offset < blockDim.x; offset *= 2) {
        float val = -HIP_INF_F;
        
        if (threadIdx.x >= offset) {
          //val = smem[threadIdx.x - offset] + smem[threadIdx.x];
          val = device_log_add(smem[threadIdx.x - offset], smem[threadIdx.x]);
        }
        
        __syncthreads();
        if (threadIdx.x >= offset) {
          smem[threadIdx.x] = val;
        }
        __syncthreads();
      }
      
      // Each thread will check to see if the sample falls in its
      // bucket
      float curBucket =
        //smem[threadIdx.x] + prevHighProb;
        device_log_add(smem[threadIdx.x], prevHighProb);
      float prevBucket =
        //threadIdx.x == 0 ? prevHighProb : smem[threadIdx.x - 1] + prevHighProb;
        threadIdx.x == 0 ? prevHighProb : device_log_add(smem[threadIdx.x - 1], prevHighProb);
      bool inBucket =
        (cat < categories) && (sample <= curBucket) && (sample > prevBucket);
      
      if (inBucket) {
        // We're done; we have the sample
        // Torch indices are 1-based
        // FIXME: broadcast exit flag?
        dest[curDist] = cat + 1;
      }
      
      // Store the previous scan's high value for future use
      //prevHighProb += smem[blockDim.x - 1];
      prevHighProb = device_log_add(prevHighProb, smem[blockDim.x - 1]);
      
      __syncthreads();
    }
  }
}

void jhu_cuda_log_sample(struct THCState *state,
                         THCudaTensor *self,
                         THCudaTensor *prob_dist) {
  
  THAssert(THCudaTensor_checkGPU(state, 2, self, prob_dist));
  if (state->rngState->current_gen == NULL) {
      THError("Random number generators have not been initialized.");
  }
  
  int inputSize = THCudaTensor_nDimension(state, prob_dist);
  THArgCheck(inputSize > 0 && inputSize <= 2, 2,
             "prob_dist must be 1 or 2 dim");
  
  // Categories are in the innermost dimension
  long numDist =
    inputSize == 1 ? 1 : THCudaTensor_size(state, prob_dist, 0);
  long numCategoriesLong =
    inputSize == 1 ? THCudaTensor_size(state, prob_dist, 0) :
    THCudaTensor_size(state, prob_dist, 1);
  
  // Since the index tensor is float, numCategories cannot exceed max
  // float integer precision
  THArgCheck(numCategoriesLong <= FLOAT32_MAX_CONSECUTIVE_INT, 2,
             "number of categories cannot exceed 2^24");
  int numCategories = (int) numCategoriesLong;
  
  // It is possible that prob_dist is non-contiguous
  THCudaTensor* probDistContig =
    THCudaTensor_newContiguous(state, prob_dist);
  
  // Restructure data for 2d
  if (inputSize == 1) {
    THCudaTensor_resize2d(state, probDistContig, 1, numCategories);
  }
  
  THCudaTensor_resize2d(state, self, numDist, 1);
  
  // Optimized allocation-free implementation
  
  // To exploit greater parallelism for the sampling, generate the
  // Uniform random samples in a separate kernel launch, into the
  // result memory. The device RNG is thread-limited
  THCudaTensor_uniform(state, self, 0.0, 1.0);
  
  hipDeviceProp_t* props = THCState_getCurrentDeviceProperties(state);
  THAssert(props != NULL);
  
  int numSM = props->multiProcessorCount;
  int maxThreads = props->maxThreadsPerBlock;
  
  dim3 block(numCategories < maxThreads ? numCategories : maxThreads);
  dim3 grid(numDist < numSM * 4 ? numDist : numSM * 4);
  
  sampleLogMultinomialOnce
    <<<grid, block, block.x * sizeof(float),
    THCState_getCurrentStream(state)>>>(
                                        THCudaTensor_data(state, self),
                                        numDist,
                                        numCategories,
                                        THCudaTensor_data(state, probDistContig));
  
  // Revert data restructuring based on input sizes
  if (inputSize == 1) {
    THCudaTensor_resize1d(state, self, 1);
    
    // Unfortunately, if prob_dist is contiguous already,
    // newContiguous is not a private copy, so we have to restructure
    // this too, so as to not affect prob_dist
    THCudaTensor_resize1d(state, probDistContig, numCategories);
  }
  
  THCudaTensor_free(state, probDistContig);
}

static int jhu_THCLogSample(lua_State *L) {
  THCState *state = getCutorchState(L);
  
  THCudaTensor *ret = (THCudaTensor*)luaT_checkudata(L, 2,
                                                     "torch.CudaTensor");
  THCudaTensor *dist = (THCudaTensor*)luaT_checkudata(L, 1,
                                                      "torch.CudaTensor");
  
  jhu_cuda_log_sample(state, ret, dist);
  
  return 0;
}

static const struct luaL_Reg jhu_THCLogSample__ [] = {
  {"logsample", jhu_THCLogSample},
  {0,0}
};

static void jhu_THCLogSample_init(lua_State *L) {
  int ret = luaT_pushmetatable(L, "torch.CudaTensor");
  if(ret == 0) {
    THError("problem pushing metatable");
  }
  luaT_registeratname(L, jhu_THCLogSample__, "jhu");
  lua_pop(L, 1);
}

#undef NUM_BLOCKS

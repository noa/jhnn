#include "hip/hip_runtime.h"
#include "THC.h"
#include "utils.h"
#include "luaT.h"

#include <lua.h>
#include <lauxlib.h>
#include <lualib.h>

#include "THCTensorRandom.h"
#include "THCDeviceUtils.cuh"
#include "THCReduceApplyUtils.cuh"
#include "THCApply.cuh"
#include "THCGeneral.h"
#include "THCTensorCopy.h"
#include "THCTensorMath.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_mtgp32_host.h>
#include <rocrand/rocrand_mtgp32_11213.h>
#include <hip/hip_math_constants.h>

#define MAX_NUM_BLOCKS 64
#define BLOCK_SIZE 256

__device__ float log_add(const float& a, const float& b) {
    if (a == -HIP_INF_F) return b;
    if (b == -HIP_INF_F) return a;
    return a>b ? a+log1p(exp(b-a)) : b+log1p(exp(a-b));
}

struct log_add_functor {
    __device__ float operator() (const float& a, const float& b) {
        return log_add(a, b);
    }
};

// convert a linear index to a row index
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T> {
    T C; // number of columns
  
    __host__ __device__
    linear_index_to_row_index(T C) : C(C) {}
    
    __host__ __device__
    T operator()(T i) {
        return i / C;
    }
};

static int jhu_THCLogSum(lua_State *L) {
    THCState *state = getCutorchState(L);
    THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 1, "torch.CudaTensor");
    THCudaTensor *output = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
    
    hipDeviceProp_t* props = THCState_getCurrentDeviceProperties(state);
    THAssert(props != NULL);
    
    long R, C;
    long ndim = THCudaTensor_nDimension(state, input);
    
    if(ndim == 2) {
        R = THCudaTensor_size(state, input, 0);
        C = THCudaTensor_size(state, input, 1);
        
        thrust::device_ptr<float> array  = thrust::device_pointer_cast(THCudaTensor_data(state, input));
        thrust::device_ptr<float> row_sums = thrust::device_pointer_cast(THCudaTensor_data(state, output));
        thrust::device_vector<int> row_indices(R);
    
        thrust::reduce_by_key
            (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(C)),
             thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(C)) + (R*C),
             array,
             row_indices.begin(),
             row_sums,
             thrust::equal_to<int>(),
             log_add_functor());
    }
    else {
        THArgCheck(0, 2, "matrix expected");
    }
    
    return 0;
}

static const struct luaL_Reg jhu_THCLogSum__ [] = {
    {"logsum", jhu_THCLogSum},
    {0, 0}
};

static void jhu_THCLogSum_init(lua_State *L) {
    int ret = luaT_pushmetatable(L, "torch.CudaTensor");
    if(ret == 0) {
        THError("problem pushing metatable");
    }
    luaT_registeratname(L, jhu_THCLogSum__, "jhu");
    lua_pop(L, 1);
}

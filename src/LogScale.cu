#include "hip/hip_runtime.h"
#include "THC.h"
#include "utils.h"
#include "luaT.h"

#include <lua.h>
#include <lauxlib.h>
#include <lualib.h>

#include "THCTensorRandom.h"
#include "THCDeviceUtils.cuh"
#include "THCReduceApplyUtils.cuh"
#include "THCApply.cuh"
#include "THCGeneral.h"
#include "THCTensorCopy.h"
#include "THCTensorMath.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_mtgp32_host.h>
#include <rocrand/rocrand_mtgp32_11213.h>
#include <hip/hip_math_constants.h>

#define MAX_NUM_BLOCKS 64
#define BLOCK_SIZE 256

__global__ void inplace_log_renorm_rows(float* dist, long rows, long cols) {
    extern __shared__ float smem[];

    for (long row = blockIdx.x; row < rows; row += gridDim.x) {
        float sum = -HIP_INF_F;
        for (long col = threadIdx.x; col < cols; col += blockDim.x) {
            sum = log_add(sum, dist[row * cols + col]);
        }
        sum = reduceBlock(smem, blockDim.x, sum, log_add_functor(), -HIP_INF_F);

        if (threadIdx.x == 0) {
            smem[0] = sum;
        }
        __syncthreads();

        sum = smem[0];

        if (sum > -HIP_INF_F) {
            for (long col = threadIdx.x; col < cols; col += blockDim.x) {
                dist[row * cols + col] = exp(dist[row * cols + col] - sum);
            }
        }
    }
}

static int jhu_THCLogScale(lua_State *L) {
  THCState *state = getCutorchState(L);
  THCudaTensor *t = (THCudaTensor*)luaT_checkudata(L, 1,
                                                   "torch.CudaTensor");

  hipDeviceProp_t* props = THCState_getCurrentDeviceProperties(state);
  THAssert(props != NULL);
  
  int numSM = props->multiProcessorCount;
  int maxThreads = props->maxThreadsPerBlock;
  
  int dim = THCudaTensor_nDimension(state, t);
  if(dim == 1) {
    long rows = 1;
    long cols = THCudaTensor_size(state, t, 0);
    
    dim3 grid(rows < numSM * 4 ? rows : numSM * 4);
    dim3 block(cols < maxThreads ? cols : maxThreads);
    
    inplace_log_renorm_rows
      <<<grid, block, block.x * sizeof(float),
      THCState_getCurrentStream(state)>>>(THCudaTensor_data(state, t),
                                          rows, cols);
  } else if(dim ==2) {
    long rows = THCudaTensor_size(state, t, 0);
    long cols = THCudaTensor_size(state, t, 1);
    
    dim3 grid(rows < numSM * 4 ? rows : numSM * 4);
    dim3 block(cols < maxThreads ? cols : maxThreads);
    
    inplace_log_renorm_rows
      <<<grid, block, block.x * sizeof(float),
      THCState_getCurrentStream(state)>>>(THCudaTensor_data(state, t),
                                          rows, cols);
  } else {
    THArgCheck(0, 2, "vector or matrix expected");
  }
  
  return 0;
}

static const struct luaL_Reg jhu_THCLogScale__ [] = {
  {"logscale", jhu_THCLogScale},
  {0,0}
};

static void jhu_THCLogScale_init(lua_State *L) {
  int ret = luaT_pushmetatable(L, "torch.CudaTensor");
  if(ret == 0) {
    THError("problem pushing metatable");
  }
  luaT_registeratname(L, jhu_THCLogScale__, "jhu");
  lua_pop(L, 1);
}

#undef NUM_BLOCKS

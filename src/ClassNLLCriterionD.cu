#include "hip/hip_runtime.h"
/**
 *  Modified from the cunn package.
 *
 * Accumulated the result as double precition, and scales it by alpha before
 * conversting to single precision.
 */
/**
 * Copyright 2014 Facebook
 */

#include "utils.h"
#include <assert.h>

static const int NTHREADS = 32;

__global__ void jhu_ClassNLLCriterionD_updateOutput_kernel1(float *output,
                                                            float *input,
                                                            float *target,
                                                            int ntarget) {
  assert(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0);

  // TODO: T4951791 Reuse code between updateOutput_kernel1 and
  // updateOutput_kernel.
  // Verify whether `register` does anything here.
  register int i, t;
  for (i = 0; i < ntarget; i++) {
    t = target[i] - 1;
    if (t >= 0)
      *output = -input[t];
  }
}

__global__ void jhu_ClassNLLCriterionD_updateOutput_kernel(float *output,
                                                           float *input,
                                                           float *target,
                                                           int nframe,
                                                           int ndim,
                                                           int sizeAverage,
                                                           int ntarget,
                                                           double alpha) {
  __shared__ double shInputs[NTHREADS];
  // Verify whether `register` does anything here.
  register int i, j, t;

  shInputs[threadIdx.x] = .0;
  for (i = threadIdx.x; i < nframe; i += NTHREADS) {
    for (j = 0; j < ntarget; ++j) {
      t = (int)target[i * ntarget + j] - 1;
      if (t >= 0)
        shInputs[threadIdx.x] += input[i * ndim + t];
    }
  }
  __syncthreads();

  // TODO: T4951791 Reuse code between updateOutput_kernel1 and
  // updateOutput_kernel
  if (threadIdx.x == 0) {
    *output = .0;
    for (i = 1; i < NTHREADS; ++i)
      shInputs[0] += shInputs[i];
    if (alpha > 0)
      *output = shInputs[0] * alpha;
    else
      *output = shInputs[0];
    if (sizeAverage)
      *output /= nframe;
    *output = -(*output);
    /*
    *output = .0;
    for (i = 0; i < NTHREADS; ++i)
      *output += shInputs[i];
    if (sizeAverage)
      *output /= nframe;
    *output = -(*output);
    */
  }
}

__global__ void jhu_ClassNLLCriterionD_updateGradInput_kernel(float *gradInput,
                                                              float *target,
                                                              int nframe,
                                                              int ndim,
                                                              float grad,
                                                              int ntarget) {
  register int i, j, t;
  for (i = threadIdx.x; i < nframe; i += NTHREADS) {
    for (j = 0; j < ntarget; ++j) {
      t = (int)target[i * ntarget + j] - 1;
      if (t >= 0)
        gradInput[i * ndim + t] = grad;
    }
  }
}

static int jhu_ClassNLLCriterionD_updateOutput(lua_State *L) {
  THCState *state = getCutorchState(L);
  THCudaTensor *input =
      (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  input = THCudaTensor_newContiguous(state, input);
  float *input_data = THCudaTensor_data(state, input);

  THCudaTensor *target =
      (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  target = THCudaTensor_newContiguous(state, target);
  float *target_data = THCudaTensor_data(state, target);
  int ntarget = 1;
  if (target->nDimension > 1)
    ntarget = target->size[1];

  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(
      L, 1, "outputTensor", "torch.CudaTensor");
  output = THCudaTensor_newContiguous(state, output);
  float *output_data = THCudaTensor_data(state, output);

  double alpha = luaT_getfieldchecknumber(L, 1, "alpha");

  if (input->nDimension == 1) {
    jhu_ClassNLLCriterionD_updateOutput_kernel1 << <1, 1>>>
        (output_data, input_data, target_data, ntarget);
  } else if (input->nDimension == 2) {
    dim3 blocks(1);
    dim3 threads(NTHREADS);
    int sizeAverage = luaT_getfieldcheckboolean(L, 1, "sizeAverage");
    jhu_ClassNLLCriterionD_updateOutput_kernel <<<blocks, threads>>>
        (output_data,
         input_data,
         target_data,
         input->size[0],
         input->size[1],
         sizeAverage,
         ntarget,
         alpha);
  } else
    THArgCheck(0, 2, "vector or matrix expected");

  hipError_t errcode = hipGetLastError();
  if (errcode != hipSuccess)
    THError(hipGetErrorString(errcode));

  THCudaTensor_free(state, output);
  THCudaTensor_free(state, target);
  THCudaTensor_free(state, input);

  return 1;
}

static int jhu_ClassNLLCriterionD_updateGradInput(lua_State *L) {
  THCState *state = getCutorchState(L);

  THCudaTensor *input =
      (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  input = THCudaTensor_newContiguous(state, input);

  THCudaTensor *target =
      (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  target = THCudaTensor_newContiguous(state, target);
  float *target_data = THCudaTensor_data(state, target);
  int ntarget = 1;
  if (target->nDimension > 1)
    ntarget = target->size[1];

  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(
      L, 1, "gradInput", "torch.CudaTensor");
  gradInput = THCudaTensor_newContiguous(state, gradInput);
  float *gradInput_data = THCudaTensor_data(state, gradInput);

  float grad = -1.0;
  if (input->nDimension == 1) {
    if (ntarget > 1)
      THArgCheck(0, 2, "multi-target not implemented");
    float tid;
    hipMemcpy(&tid, target_data, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(gradInput_data + (int)tid - 1,
               &grad,
               sizeof(float),
               hipMemcpyHostToDevice);
  } else if (input->nDimension == 2) {
    int nframe = input->size[0];
    int ndim = input->size[1];
    int sizeAverage = luaT_getfieldcheckboolean(L, 1, "sizeAverage");
    if (sizeAverage)
      grad /= nframe;
    dim3 blocks(1);
    dim3 threads(NTHREADS);
    jhu_ClassNLLCriterionD_updateGradInput_kernel <<<blocks, threads>>>
        (gradInput_data, target_data, nframe, ndim, grad, ntarget);
  } else
    THArgCheck(0, 2, "vector or matrix expected");

  hipError_t errcode = hipGetLastError();
  if (errcode != hipSuccess)
    THError(hipGetErrorString(errcode));

  THCudaTensor_free(state, gradInput);
  THCudaTensor_free(state, target);
  THCudaTensor_free(state, input);

  return 1;
}

static const struct luaL_Reg jhu_ClassNLLCriterionD__[] = {
    {"ClassNLLCriterionD_updateOutput", jhu_ClassNLLCriterionD_updateOutput},
    {"ClassNLLCriterionD_updateGradInput",
     jhu_ClassNLLCriterionD_updateGradInput},
    {NULL, NULL}};

void jhu_ClassNLLCriterionD_init(lua_State *L) {
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, jhu_ClassNLLCriterionD__, "cujhu");
  lua_pop(L, 1);
}

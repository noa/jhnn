#include "hip/hip_runtime.h"
#include "THC.h"
#include "utils.h"
#include "luaT.h"

#include <lua.h>
#include <lauxlib.h>
#include <lualib.h>

#include "THCTensorRandom.h"
#include "THCDeviceUtils.cuh"
#include "THCReduceApplyUtils.cuh"
#include "THCApply.cuh"
#include "THCGeneral.h"
#include "THCTensorCopy.h"
#include "THCTensorMath.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_mtgp32_host.h>
#include <rocrand/rocrand_mtgp32_11213.h>
#include <hip/hip_math_constants.h>

#define MAX_NUM_BLOCKS 64
#define BLOCK_SIZE 256

__global__ void inplace_renorm_rows(float* dist, long rows, long cols) {
    extern __shared__ float smem[];

    for (long row = blockIdx.x; row < rows; row += gridDim.x) {
        float sum = 0;
        for (long col = threadIdx.x; col < cols; col += blockDim.x) {
            sum += dist[row * cols + col];
        }
        sum = reduceBlock(smem, blockDim.x, sum, thrust::plus<float>(), 0.0f);

        if (threadIdx.x == 0) {
            smem[0] = sum;
        }
        __syncthreads();

        sum = smem[0];

        if (sum > 0.0f) {
            for (long col = threadIdx.x; col < cols; col += blockDim.x) {
                dist[row * cols + col] = dist[row * cols + col]/sum;
            }
        }
    }
}

static int jhu_THCScale(lua_State *L) {
  THCState *state = getCutorchState(L);
  THCudaTensor *t = (THCudaTensor*)luaT_checkudata(L, 1,
                                                   "torch.CudaTensor");

  hipDeviceProp_t* props = THCState_getCurrentDeviceProperties(state);
  THAssert(props != NULL);
  
  int numSM = props->multiProcessorCount;
  int maxThreads = props->maxThreadsPerBlock;
  
  int dim = THCudaTensor_nDimension(state, t);
  if(dim == 1) {
    long rows = 1;
    long cols = THCudaTensor_size(state, t, 0);
    
    dim3 grid(rows < numSM * 4 ? rows : numSM * 4);
    dim3 block(cols < maxThreads ? cols : maxThreads);
    
    inplace_renorm_rows
      <<<grid, block, block.x * sizeof(float),
      THCState_getCurrentStream(state)>>>(THCudaTensor_data(state, t),
                                          rows, cols);
  } else if(dim ==2) {
    long rows = THCudaTensor_size(state, t, 0);
    long cols = THCudaTensor_size(state, t, 1);
    
    dim3 grid(rows < numSM * 4 ? rows : numSM * 4);
    dim3 block(cols < maxThreads ? cols : maxThreads);
    
    inplace_renorm_rows
      <<<grid, block, block.x * sizeof(float),
      THCState_getCurrentStream(state)>>>(THCudaTensor_data(state, t),
                                          rows, cols);
  } else {
    THArgCheck(0, 2, "vector or matrix expected");
  }
  
  return 0;
}

static const struct luaL_Reg jhu_THCScale__ [] = {
  {"scale", jhu_THCScale},
  {0,0}
};

static void jhu_THCScale_init(lua_State *L) {
  int ret = luaT_pushmetatable(L, "torch.CudaTensor");
  if(ret == 0) {
    THError("problem pushing metatable");
  }
  luaT_registeratname(L, jhu_THCScale__, "jhu");
  lua_pop(L, 1);
}

#undef NUM_BLOCKS
